#include "hip/hip_runtime.h"

#include "gtest/gtest.h"

#include "cstone/gravity/treewalk.hpp"

#include "ryoanji/dataset.h"
#include "ryoanji/direct.cuh"

std::vector<fvec4> cpuReference(const std::vector<fvec4>& bodies)
{
    size_t numBodies = bodies.size();

    std::vector<double> x(numBodies);
    std::vector<double> y(numBodies);
    std::vector<double> z(numBodies);
    std::vector<double> h(numBodies, 0.0);
    std::vector<double> m(numBodies);

    for (size_t i = 0; i < numBodies; ++i)
    {
        x[i] = bodies[i][0];
        y[i] = bodies[i][1];
        z[i] = bodies[i][2];
        m[i] = bodies[i][3];
    }

    std::vector<double> ax(numBodies);
    std::vector<double> ay(numBodies);
    std::vector<double> az(numBodies);
    std::vector<double> pot(numBodies);

    cstone::directSum(x.data(), y.data(), z.data(), h.data(), m.data(), numBodies,
                      ax.data(), ay.data(), az.data(), pot.data());

    std::vector<fvec4> acc(numBodies, fvec4(0));

    for (size_t i = 0; i < numBodies; ++i)
    {
        acc[i] = fvec4(pot[i], ax[i], ay[i], az[i]);
    }

    return acc;
}

TEST(DirectSum, MatchCpu)
{
    int npOnEdge  = 10;
    int numBodies = npOnEdge * npOnEdge * npOnEdge;

    // the CPU reference uses mass softening, while the GPU P2P kernel still uses plummer softening
    // so the only way to compare is without softening in both versions and make sure that
    // particles are not on top of each other
    float eps = 0.0;

    auto bodies = makeGridBodies(npOnEdge, 0.5);

    cudaVec<fvec4> bodyPos(numBodies, true);
    for (size_t i = 0; i < numBodies; ++i)
    {
        bodyPos[i] = bodies[i];
        //printf("%f %f %f %f\n", bodyPos[i][0], bodyPos[i][1], bodyPos[i][2], bodyPos[i][3]);
    }
    bodyPos.h2d();

    cudaVec<fvec4> bodyAcc(numBodies, true);
    bodyAcc.zeros();

    directSum(eps, bodyPos, bodyAcc);

    bodyAcc.d2h();

    auto refAcc = cpuReference(bodies);

    for (int i = 0; i < numBodies; ++i)
    {
        fvec3 ref   = {refAcc[i][1], refAcc[i][2], refAcc[i][3]};
        fvec3 probe = {bodyAcc[i][1], bodyAcc[i][2], bodyAcc[i][3]};

        EXPECT_NEAR(std::sqrt(norm(ref-probe)/norm(probe)), 0, 1e-6);
        // the potential
        EXPECT_NEAR(refAcc[i][0], bodyAcc[i][0], 1e-6);

        //printf("%f %f %f\n", ref[1], ref[2], ref[3]);
        //printf("%f %f %f\n", probe[1], probe[2], probe[3]);
    }
}
