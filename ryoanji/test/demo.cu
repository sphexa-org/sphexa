#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2024 CSCS, ETH Zurich, University of Basel, University of Zurich
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Single-GPU demonstrator app for the Ryoanji N-body library
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 * @author Rio Yokota <rioyokota@gsic.titech.ac.jp>
 */

#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/cuda/gpu_config.cuh"
#include "cstone/cuda/thrust_util.cuh"
#include "cstone/focus/source_center_gpu.h"
#include "cstone/traversal/groups_gpu.h"
#include "cstone/util/array.hpp"

#include "nbody/dataset.hpp"
#include "ryoanji/interface/treebuilder.cuh"
#include "ryoanji/nbody/types.h"
#include "ryoanji/nbody/traversal.cuh"
#include "ryoanji/nbody/direct.cuh"
#include "ryoanji/nbody/upsweep_gpu.h"

using namespace ryoanji;

template<class Tc, class Th, class Tm, class Ta, class Tf, class MType>
util::array<Tc, 5> computeAcceleration(size_t firstBody, size_t lastBody, const Tc* x, const Tc* y, const Tc* z,
                                       const Tm* m, const Th* h, Tc G, int numShells, const cstone::Box<Tc>& box, Ta* p,
                                       Ta* ax, Tc* ay, Tc* az, const TreeNodeIndex* childOffsets,
                                       const TreeNodeIndex* internalToLeaf, const LocalIndex* layout,
                                       const Vec4<Tf>* sourceCenter, const MType* Multipole);

template<class KeyType, class T, class MType>
void upsweep(int numSources, int numLeaves, int numLevels, float theta, const TreeNodeIndex* levelRange, const T* x,
             const T* y, const T* z, const T* m, const cstone::Box<T>& box, const LocalIndex* layout,
             const KeyType* prefixes, const TreeNodeIndex* childOffsets, const TreeNodeIndex* leafToInternal,
             Vec4<T>* centers, MType* Multipole);

int main(int argc, char** argv)
{
    using T             = float;
    using MultipoleType = CartesianQuadrupole<T>;

    int power     = argc > 1 ? std::stoi(argv[1]) : 17;
    int directRef = argc > 2 ? std::stoi(argv[2]) : 1;
    int numShells = argc > 3 ? std::stoi(argv[3]) : 0;

    std::size_t numBodies = (1 << power) - 1;
    T           theta     = 0.6;
    T           boxSize   = 3;
    T           G         = 1.0;

    const int ncrit = 64;

    fprintf(stdout, "--- BH Parameters ---------------\n");
    fprintf(stdout, "numBodies            : %lu\n", numBodies);
    fprintf(stdout, "theta                : %f\n", theta);
    fprintf(stdout, "ncrit                : %d\n", ncrit);

    thrust::host_vector<T> x(numBodies), y(numBodies), z(numBodies), m(numBodies), h(numBodies);
    makeCubeBodies(x.data(), y.data(), z.data(), m.data(), h.data(), numBodies, boxSize);

    // upload bodies to device
    thrust::device_vector<T> d_x = x, d_y = y, d_z = z, d_m = m, d_h = h;

    cstone::Box<T> box(-boxSize, boxSize);

    TreeBuilder<uint64_t> treeBuilder(ncrit);
    int                   numSources = treeBuilder.update(rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), numBodies, box);

    const TreeNodeIndex* levelRange   = treeBuilder.levelRange();
    int                  highestLevel = treeBuilder.maxTreeLevel();

    thrust::device_vector<Vec4<T>>       sourceCenter(numSources);
    thrust::device_vector<MultipoleType> Multipole(numSources);

    upsweep(numSources, treeBuilder.numLeafNodes(), highestLevel, theta, levelRange, rawPtr(d_x), rawPtr(d_y),
            rawPtr(d_z), rawPtr(d_m), box, treeBuilder.layout(), treeBuilder.nodeKeys(), treeBuilder.childOffsets(),
            treeBuilder.leafToInternal(), rawPtr(sourceCenter), rawPtr(Multipole));

    thrust::device_vector<T> d_p(numBodies, 0), d_ax(numBodies, 0), d_ay(numBodies, 0), d_az(numBodies, 0);

    fprintf(stdout, "--- BH Profiling ----------------\n");

    auto t0 = std::chrono::high_resolution_clock::now();

    auto interactions = computeAcceleration(0, numBodies, rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_m),
                                            rawPtr(d_h), G, numShells, box, rawPtr(d_p), rawPtr(d_ax), rawPtr(d_ay),
                                            rawPtr(d_az), treeBuilder.childOffsets(), treeBuilder.internalToLeaf(),
                                            treeBuilder.layout(), rawPtr(sourceCenter), rawPtr(Multipole));

    auto   t1    = std::chrono::high_resolution_clock::now();
    double dt    = std::chrono::duration<double>(t1 - t0).count();
    double flops = (interactions[0] * 23 + interactions[2] * 65) * numBodies / dt / 1e12;

    fprintf(stdout, "--- Total runtime ----------------\n");
    fprintf(stdout, "Total BH            : %.7f s (%.7f TFlops)\n", dt, flops);

    if (!directRef) { return 0; }

    thrust::device_vector<T> refP(numBodies), refAx(numBodies), refAy(numBodies), refAz(numBodies);

    t0 = std::chrono::high_resolution_clock::now();
    directSum(0, numBodies, numBodies, Vec3<T>{box.lx(), box.ly(), box.lz()}, numShells, rawPtr(d_x), rawPtr(d_y),
              rawPtr(d_z), rawPtr(d_m), rawPtr(d_h), rawPtr(refP), rawPtr(refAx), rawPtr(refAy), rawPtr(refAz));

    t1 = std::chrono::high_resolution_clock::now();
    dt = std::chrono::duration<double>(t1 - t0).count();

    flops = std::pow((2 * numShells + 1), 3) * 23. * numBodies * numBodies / dt / 1e12;
    fprintf(stdout, "Total Direct         : %.7f s (%.7f TFlops)\n", dt, flops);

    thrust::host_vector<T> h_p  = d_p;
    thrust::host_vector<T> h_ax = d_ax;
    thrust::host_vector<T> h_ay = d_ay;
    thrust::host_vector<T> h_az = d_az;

    double                 referencePotential = 0.5 * G * thrust::reduce(refP.begin(), refP.end(), 0.0);
    thrust::host_vector<T> h_refAx            = refAx;
    thrust::host_vector<T> h_refAy            = refAy;
    thrust::host_vector<T> h_refAz            = refAz;

    std::vector<double> delta(numBodies);

    double potentialSum = 0;
    for (int i = 0; i < numBodies; i++)
    {
        potentialSum += h_p[i];
        Vec3<T> ref   = {h_refAx[i], h_refAy[i], h_refAz[i]};
        Vec3<T> probe = {h_ax[i], h_ay[i], h_az[i]};
        delta[i]      = std::sqrt(norm2(ref - probe) / norm2(ref));
    }

    std::sort(begin(delta), end(delta));

    fprintf(stdout, "--- BH vs. direct ---------------\n");

    std::cout << "potentials, body-sum: " << 0.5 * G * potentialSum << " atomic sum: " << 0.5 * G * interactions[4]
              << " reference: " << referencePotential << std::endl;
    std::cout << "min Error: " << delta[0] << std::endl;
    std::cout << "50th percentile: " << delta[numBodies / 2] << std::endl;
    std::cout << "10th percentile: " << delta[numBodies * 0.9] << std::endl;
    std::cout << "1st percentile: " << delta[numBodies * 0.99] << std::endl;
    std::cout << "max Error: " << delta[numBodies - 1] << std::endl;

    fprintf(stdout, "--- Tree stats -------------------\n");
    fprintf(stdout, "Bodies               : %lu\n", numBodies);
    fprintf(stdout, "Cells                : %d\n", numSources);
    fprintf(stdout, "Tree depth           : %d\n", highestLevel);
    fprintf(stdout, "--- Traversal stats --------------\n");
    fprintf(stdout, "P2P mean list length : %d (max %d)\n", int(interactions[0]), int(interactions[1]));
    fprintf(stdout, "M2P mean list length : %d (max %d)\n", int(interactions[2]), int(interactions[3]));

    return 0;
}

/*! @brief Compute approximate body accelerations with Barnes-Hut
 *
 * @param[in]    firstBody      index of first body in @p bodyPos to compute acceleration for
 * @param[in]    lastBody       index (exclusive) of last body in @p bodyPos to compute acceleration for
 * @param[in]    x,y,z,m,h      bodies, in SFC order and as referenced by sourceCells
 * @param[in]    G              gravitational constant
 * @param[in]    numShells      number of periodic shells in each dimension to include
 * @param[in]    box            coordinate bounding box
 * @param[inout] p              body potential to add to, on device
 * @param[inout] ax,ay,az       body acceleration to add to
 * @param[in]    childOffsets   location (index in [0:numTreeNodes]) of first child of each cell, 0 indicates a leaf
 * @param[in]    internalToLeaf for each cell in [0:numTreeNodes], stores the leaf cell (cstone) index in [0:numLeaves]
 *                              if the cell is not a leaf, the value is negative
 * @param[in]    layout         for each leaf cell in [0:numLeaves], stores the index of the first body in the cell
 * @param[in]    sourceCenter   x,y,z center and square MAC radius of each cell in [0:numTreeNodes]
 * @param[in]    Multipole      cell multipoles, on device
 * @return                      P2P and M2P interaction statistics
 */
template<class Tc, class Th, class Tm, class Ta, class Tf, class MType>
util::array<Tc, 5> computeAcceleration(size_t firstBody, size_t lastBody, const Tc* x, const Tc* y, const Tc* z,
                                       const Tm* m, const Th* h, Tc G, int numShells, const cstone::Box<Tc>& box, Ta* p,
                                       Ta* ax, Tc* ay, Tc* az, const TreeNodeIndex* childOffsets,
                                       const TreeNodeIndex* internalToLeaf, const LocalIndex* layout,
                                       const Vec4<Tf>* sourceCenter, const MType* Multipole)
{
    constexpr int numWarpsPerBlock = TravConfig::numThreads / GpuConfig::warpSize;

    cstone::GroupData<cstone::GpuTag> groups;
    cstone::computeFixedGroups(firstBody, lastBody, TravConfig::targetSize, groups);

    LocalIndex numBodies  = lastBody - firstBody;
    int        numTargets = (numBodies - 1) / TravConfig::targetSize + 1;
    int        numBlocks  = (numTargets - 1) / numWarpsPerBlock + 1;
    numBlocks             = std::min(numBlocks, TravConfig::maxNumActiveBlocks);

    printf("launching %d blocks\n", numBlocks);

    const int                  poolSize = TravConfig::memPerWarp * numWarpsPerBlock * numBlocks;
    thrust::device_vector<int> globalPool(poolSize);

    resetTraversalCounters<<<1, 1>>>();
    traverse<<<numBlocks, TravConfig::numThreads>>>(
        groups.view(), 1, x, y, z, m, h, childOffsets, internalToLeaf, layout, sourceCenter, Multipole, G, numShells,
        {box.lx(), box.ly(), box.lz()}, p, ax, ay, az, thrust::raw_pointer_cast(globalPool.data()));
    kernelSuccess("traverse");

    typename BhStats::type stats[BhStats::numStats];
    checkGpuErrors(hipMemcpyFromSymbol(stats, HIP_SYMBOL(GPU_SYMBOL(bhStats)), BhStats::numStats * sizeof(BhStats::type)));

    auto sumP2P = stats[BhStats::sumP2P];
    auto maxP2P = stats[BhStats::maxP2P];
    auto sumM2P = stats[BhStats::sumM2P];
    auto maxM2P = stats[BhStats::maxM2P];

    float totalPotential;
    checkGpuErrors(hipMemcpyFromSymbol(&totalPotential, HIP_SYMBOL(GPU_SYMBOL(totalPotentialGlob)), sizeof(float)));

    util::array<Tc, 5> interactions;
    interactions[0] = Tc(sumP2P) / Tc(numBodies);
    interactions[1] = Tc(maxP2P);
    interactions[2] = Tc(sumM2P) / Tc(numBodies);
    interactions[3] = Tc(maxM2P);
    interactions[4] = totalPotential;

    return interactions;
}

template<class KeyType, class T, class MType>
void upsweep(int numSources, int numLeaves, int numLevels, float theta, const TreeNodeIndex* levelRange, const T* x,
             const T* y, const T* z, const T* m, const cstone::Box<T>& box, const LocalIndex* layout,
             const KeyType* prefixes, const TreeNodeIndex* childOffsets, const TreeNodeIndex* leafToInternal,
             Vec4<T>* centers, MType* Multipole)
{
    auto t0 = std::chrono::high_resolution_clock::now();

    cstone::computeLeafSourceCenterGpu(x, y, z, m, leafToInternal, numLeaves, layout, centers);
    cstone::upsweepCentersGpu(cstone::maxTreeLevel<KeyType>{}, levelRange, childOffsets, centers);

    computeLeafMultipoles(x, y, z, m, leafToInternal, numLeaves, layout, centers, Multipole);
    for (int level = numLevels - 1; level >= 1; level--)
    {
        upsweepMultipoles(levelRange[level], levelRange[level + 1], childOffsets, centers, Multipole);
    }

    cstone::setMacGpu(prefixes, numSources, centers, 1.f / theta, box);

    auto   t1 = std::chrono::high_resolution_clock::now();
    double dt = std::chrono::duration<double>(t1 - t0).count();

    fprintf(stdout, "Upward pass          : %.7f s\n", dt);
}
