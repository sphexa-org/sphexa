#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Integral-approach-to-derivative i-loop GPU driver
 *
 * @author Ruben Cabezon <ruben.cabezon@unibas.ch>
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_std/iad_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

/*! @brief
 *
 * @tparam     T               float or double
 * @tparam     KeyType         32- or 64-bit unsigned integer
 * @param[in]  sincIndex
 * @param[in]  K
 * @param[in]  ngmax           maximum number of neighbors per particle to use
 * @param[in]  box             global coordinate bounding box
 * @param[in]  firstParticle   first particle to compute
 * @param[in]  lastParticle    last particle to compute
 * @param[in]  numParticles    number of local particles + halos
 * @param[in]  particleKeys    SFC keys of particles, sorted in ascending order
 * @param[in]  x               x coords, length @p numParticles, SFC sorted
 * @param[in]  y               y coords, length @p numParticles, SFC sorted
 * @param[in]  z               z coords, length @p numParticles, SFC sorted
 * @param[in]  h               smoothing lengths, length @p numParticles
 * @param[in]  m               masses, length @p numParticles
 * @param[in]  rho             densities, length @p numParticles
 * @param[in]  wh              sinc lookup table
 * @param[in]  whd             sinc derivative lookup table
 * @param[out] c11             output IAD components, length @p numParticles
 * @param[out] c12
 * @param[out] c13
 * @param[out] c22
 * @param[out] c23
 * @param[out] c33
 */
template<class Tc, class Tm, class T, class KeyType>
__global__ void IADGpuKernel(T sincIndex, T K, unsigned ngmax, cstone::Box<T> box, size_t first, size_t last,
                             const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y, const Tc* z,
                             const T* h, const Tm* m, const T* rho, const T* wh, const T* whd, T* c11, T* c12, T* c13,
                             T* c22, T* c23, T* c33, LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned numTargets  = (last - first - 1) / TravConfig::targetSize + 1;
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    cstone::LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numTargets) { return; }

        cstone::LocalIndex bodyBegin = first + targetIdx * TravConfig::targetSize;
        cstone::LocalIndex bodyEnd   = cstone::imin(bodyBegin + TravConfig::targetSize, last);
        cstone::LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= last) { continue; }

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        sph::IADJLoopSTD<TravConfig::targetSize>(i, sincIndex, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, h, m,
                                                 rho, wh, whd, c11, c12, c13, c22, c23, c33);
    }
}

template<class Dataset>
void computeIADGpu(size_t startIndex, size_t endIndex, unsigned ngmax, Dataset& d,
                   const cstone::Box<typename Dataset::RealType>& box)
{
    unsigned numWarpsPerBlock = TravConfig::numThreads / GpuConfig::warpSize;
    unsigned numBodies        = endIndex - startIndex;
    unsigned numWarps         = (numBodies - 1) / TravConfig::targetSize + 1;
    unsigned numBlocks        = (numWarps - 1) / numWarpsPerBlock + 1;
    numBlocks                 = std::min(numBlocks, TravConfig::maxNumActiveBlocks);

    unsigned poolSize = TravConfig::memPerWarp * numWarpsPerBlock * numBlocks;
    unsigned nidxSize = ngmax * numBlocks * TravConfig::numThreads;
    reallocateDestructive(d.devData.traversalStack, poolSize + nidxSize, 1.01);
    auto* traversalPool = reinterpret_cast<TreeNodeIndex*>(rawPtr(d.devData.traversalStack));
    auto* nidxPool      = rawPtr(d.devData.traversalStack) + poolSize;

    cstone::resetTraversalCounters<<<1, 1>>>();

    IADGpuKernel<<<numBlocks, TravConfig::numThreads>>>(
        d.sincIndex, d.K, ngmax, box, startIndex, endIndex, d.treeView, rawPtr(d.devData.x), rawPtr(d.devData.y),
        rawPtr(d.devData.z), rawPtr(d.devData.h), rawPtr(d.devData.m), rawPtr(d.devData.rho), rawPtr(d.devData.wh),
        rawPtr(d.devData.whd), rawPtr(d.devData.c11), rawPtr(d.devData.c12), rawPtr(d.devData.c13),
        rawPtr(d.devData.c22), rawPtr(d.devData.c23), rawPtr(d.devData.c33), nidxPool, traversalPool);
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeIADGpu(size_t, size_t, unsigned, sphexa::ParticlesData<double, unsigned, cstone::GpuTag>& d,
                            const cstone::Box<double>&);
template void computeIADGpu(size_t, size_t, unsigned, sphexa::ParticlesData<double, uint64_t, cstone::GpuTag>& d,
                            const cstone::Box<double>&);
template void computeIADGpu(size_t, size_t, unsigned, sphexa::ParticlesData<float, unsigned, cstone::GpuTag>& d,
                            const cstone::Box<float>&);
template void computeIADGpu(size_t, size_t, unsigned, sphexa::ParticlesData<float, uint64_t, cstone::GpuTag>& d,
                            const cstone::Box<float>&);

} // namespace sph
