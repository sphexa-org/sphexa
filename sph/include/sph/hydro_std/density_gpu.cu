#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_std/density_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

template<class Tc, class Tm, class T, class KeyType>
__global__ void cudaDensity(T K, unsigned ng0, unsigned ngmax, cstone::Box<T> box, const cstone::LocalIndex* groups,
                            cstone::LocalIndex numGroups, const cstone::OctreeNsView<Tc, KeyType> tree, unsigned* nc,
                            const Tc* x, const Tc* y, const Tc* z, T* h, const Tm* m, const T* wh, const T* whd, T* rho,
                            LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    cstone::LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) { return; }

        cstone::LocalIndex bodyBegin = groups[targetIdx];
        cstone::LocalIndex bodyEnd   = groups[targetIdx + 1];
        cstone::LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        constexpr int ncMaxIteration = 10;
        for (int ncIt = 0; ncIt < ncMaxIteration; ++ncIt)
        {
            bool repeat = (ncTrue[0] < ng0 / 4 || ncTrue[0] > ngmax) && i < bodyEnd;
            if (!cstone::ballotSync(repeat)) { break; }
            if (repeat) h[i] = updateH(ng0, ncTrue[0], h[i]);
            ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);
        }

        if (i >= bodyEnd) { continue; }

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        rho[i] = sph::densityJLoop<TravConfig::targetSize>(i, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z, h, m,
                                                           wh, whd);
        nc[i]  = ncTrue[0];
    }
}

template<class Dataset>
void computeDensityGpu(size_t startIndex, size_t endIndex, Dataset& d,
                       const cstone::Box<typename Dataset::RealType>& box)
{
    unsigned numBodies = endIndex - startIndex;
    unsigned numBlocks = TravConfig::numBlocks(numBodies);

    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, numBodies, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    unsigned numGroups = d.devData.targetGroups.size() - 1;
    cudaDensity<<<numBlocks, TravConfig::numThreads>>>(
        d.K, d.ng0, d.ngmax, box, rawPtr(d.devData.targetGroups), numGroups, d.treeView.nsView(), rawPtr(d.devData.nc),
        rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.h), rawPtr(d.devData.m),
        rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.rho), nidxPool, traversalPool);
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeDensityGpu(size_t, size_t, sphexa::ParticlesData<double, unsigned, cstone::GpuTag>&,
                                const cstone::Box<double>&);
template void computeDensityGpu(size_t, size_t, sphexa::ParticlesData<double, uint64_t, cstone::GpuTag>&,
                                const cstone::Box<double>&);
template void computeDensityGpu(size_t, size_t, sphexa::ParticlesData<float, unsigned, cstone::GpuTag>&,
                                const cstone::Box<float>&);
template void computeDensityGpu(size_t, size_t, sphexa::ParticlesData<float, uint64_t, cstone::GpuTag>&,
                                const cstone::Box<float>&);

} // namespace sph
