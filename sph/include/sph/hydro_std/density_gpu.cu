#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Density i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <algorithm>

#include "sph/sph.cuh"
#include "sph/particles_data.hpp"
#include "sph/util/cuda_utils.cuh"
#include "sph/hydro_std/density_kern.hpp"

#include "cstone/cuda/findneighbors.cuh"

namespace sph
{
namespace cuda
{

template<class T, class KeyType>
__global__ void cudaDensity(T sincIndex, T K, int ngmax, cstone::Box<T> box, int firstParticle, int lastParticle,
                            int numParticles, const KeyType* particleKeys, int* neighborsCount, const T* x, const T* y,
                            const T* z, const T* h, const T* m, const T* wh, const T* whd, T* rho)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i   = tid + firstParticle;
    if (i >= lastParticle) return;

    // need to hard-code ngmax stack allocation for now
    assert(ngmax <= NGMAX && "ngmax too big, please increase NGMAX to desired value");
    int neighbors[NGMAX];
    int neighborsCount_;

    // starting from CUDA 11.3, dynamic stack allocation is available with the following command
    // int* neighbors = (int*)alloca(ngmax * sizeof(int));

    cstone::findNeighbors(
        i, x, y, z, h, box, cstone::sfcKindPointer(particleKeys), neighbors, &neighborsCount_, numParticles, ngmax);

    rho[i] = sph::densityJLoop(i, sincIndex, K, box, neighbors, neighborsCount_, x, y, z, h, m, wh, whd);

    neighborsCount[tid] = neighborsCount_;
}

template<class Dataset>
void computeDensity(size_t startIndex, size_t endIndex, size_t ngmax, Dataset& d,
                    const cstone::Box<typename Dataset::RealType>& box)
{
    using T       = typename Dataset::RealType;
    using KeyType = typename Dataset::KeyType;

    size_t sizeWithHalos     = d.x.size();
    size_t numLocalParticles = endIndex - startIndex;

    size_t taskSize = sphexa::DeviceParticlesData<T, KeyType>::taskSize;
    size_t numTasks = iceil(numLocalParticles, taskSize);

    // number of CUDA streams to use
    constexpr int NST = sphexa::DeviceParticlesData<T, Dataset>::NST;

    for (int i = 0; i < numTasks; ++i)
    {
        int          sIdx   = i % NST;
        hipStream_t stream = d.devData.d_stream[sIdx].stream;

        int* d_neighborsCount_use = d.devData.d_stream[sIdx].d_neighborsCount;

        unsigned firstParticle       = startIndex + i * taskSize;
        unsigned lastParticle        = std::min(startIndex + (i + 1) * taskSize, endIndex);
        unsigned numParticlesCompute = lastParticle - firstParticle;

        unsigned numThreads = 256;
        unsigned numBlocks  = (numParticlesCompute + numThreads - 1) / numThreads;

        cudaDensity<<<numBlocks, numThreads, 0, stream>>>(d.sincIndex,
                                                          d.K,
                                                          ngmax,
                                                          box,
                                                          firstParticle,
                                                          lastParticle,
                                                          sizeWithHalos,
                                                          rawPtr(d.devData.codes),
                                                          d_neighborsCount_use,
                                                          rawPtr(d.devData.x),
                                                          rawPtr(d.devData.y),
                                                          rawPtr(d.devData.z),
                                                          rawPtr(d.devData.h),
                                                          rawPtr(d.devData.m),
                                                          rawPtr(d.devData.wh),
                                                          rawPtr(d.devData.whd),
                                                          rawPtr(d.devData.rho));
        CHECK_CUDA_ERR(hipGetLastError());

        CHECK_CUDA_ERR(hipMemcpyAsync(d.neighborsCount.data() + firstParticle,
                                       d_neighborsCount_use,
                                       numParticlesCompute * sizeof(decltype(d.neighborsCount.front())),
                                       hipMemcpyDeviceToHost,
                                       stream));
    }
}

template void computeDensity(size_t, size_t, size_t, sphexa::ParticlesData<double, unsigned, cstone::GpuTag>&,
                             const cstone::Box<double>&);
template void computeDensity(size_t, size_t, size_t, sphexa::ParticlesData<double, uint64_t, cstone::GpuTag>&,
                             const cstone::Box<double>&);
template void computeDensity(size_t, size_t, size_t, sphexa::ParticlesData<float, unsigned, cstone::GpuTag>&,
                             const cstone::Box<float>&);
template void computeDensity(size_t, size_t, size_t, sphexa::ParticlesData<float, uint64_t, cstone::GpuTag>&,
                             const cstone::Box<float>&);

} // namespace cuda
} // namespace sph
