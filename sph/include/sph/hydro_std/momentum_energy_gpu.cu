#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Pressure gradient (momentum) and energy i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <hipcub/hipcub.hpp>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/util/device_math.cuh"
#include "sph/hydro_std/momentum_energy_kern.hpp"

namespace sph
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

static __device__ float minDt_device;

template<class Tc, class Tm, class T, class Tm1, class KeyType>
__global__ void cudaGradP(T sincIndex, T K, T Kcour, unsigned ngmax, cstone::Box<T> box, size_t first, size_t last,
                          const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y, const Tc* z,
                          const T* vx, const T* vy, const T* vz, const T* h, const Tm* m, const T* rho, const T* p,
                          const T* c, const T* c11, const T* c12, const T* c13, const T* c22, const T* c23,
                          const T* c33, const T* wh, const T* whd, T* grad_P_x, T* grad_P_y, T* grad_P_z, Tm1* du,
                          LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned numTargets  = (last - first - 1) / TravConfig::targetSize + 1;
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    cstone::LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    T dt_i = INFINITY;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numTargets) { break; }

        cstone::LocalIndex bodyBegin = first + targetIdx * TravConfig::targetSize;
        cstone::LocalIndex bodyEnd   = cstone::imin(bodyBegin + TravConfig::targetSize, last);
        cstone::LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= last) continue;

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        T        maxvsignal;

        momentumAndEnergyJLoop<TravConfig::targetSize>(i, sincIndex, K, box, neighborsWarp + laneIdx, ncCapped, x, y, z,
                                                       vx, vy, vz, h, m, rho, p, c, c11, c12, c13, c22, c23, c33, wh,
                                                       whd, grad_P_x, grad_P_y, grad_P_z, du, &maxvsignal);

        dt_i = stl::min(dt_i, tsKCourant(maxvsignal, h[i], c[i], Kcour));
    }

    typedef hipcub::BlockReduce<T, TravConfig::numThreads> BlockReduce;
    __shared__ typename BlockReduce::TempStorage        temp_storage;

    BlockReduce reduce(temp_storage);
    T           blockMin = reduce.Reduce(dt_i, hipcub::Min());
    __syncthreads();

    if (threadIdx.x == 0) { atomicMinFloat(&minDt_device, blockMin); }
}

template<class Dataset>
void computeMomentumEnergyStdGpu(size_t startIndex, size_t endIndex, Dataset& d,
                                 const cstone::Box<typename Dataset::RealType>& box)
{
    unsigned numWarpsPerBlock = TravConfig::numThreads / GpuConfig::warpSize;
    unsigned numBodies        = endIndex - startIndex;
    unsigned numWarps         = (numBodies - 1) / TravConfig::targetSize + 1;
    unsigned numBlocks        = (numWarps - 1) / numWarpsPerBlock + 1;
    numBlocks                 = std::min(numBlocks, TravConfig::maxNumActiveBlocks);

    unsigned poolSize = TravConfig::memPerWarp * numWarpsPerBlock * numBlocks;
    unsigned nidxSize = d.ngmax * numBlocks * TravConfig::numThreads;
    reallocateDestructive(d.devData.traversalStack, poolSize + nidxSize, 1.01);
    auto* traversalPool = reinterpret_cast<TreeNodeIndex*>(rawPtr(d.devData.traversalStack));
    auto* nidxPool      = rawPtr(d.devData.traversalStack) + poolSize;

    float huge = 1e10;
    checkGpuErrors(hipMemcpyToSymbol(HIP_SYMBOL(minDt_device), &huge, sizeof(huge)));
    cstone::resetTraversalCounters<<<1, 1>>>();

    cudaGradP<<<numBlocks, TravConfig::numThreads>>>(
        d.sincIndex, d.K, d.Kcour, d.ngmax, box, startIndex, endIndex, d.treeView, rawPtr(d.devData.x),
        rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.vx), rawPtr(d.devData.vy), rawPtr(d.devData.vz),
        rawPtr(d.devData.h), rawPtr(d.devData.m), rawPtr(d.devData.rho), rawPtr(d.devData.p), rawPtr(d.devData.c),
        rawPtr(d.devData.c11), rawPtr(d.devData.c12), rawPtr(d.devData.c13), rawPtr(d.devData.c22),
        rawPtr(d.devData.c23), rawPtr(d.devData.c33), rawPtr(d.devData.wh), rawPtr(d.devData.whd), rawPtr(d.devData.ax),
        rawPtr(d.devData.ay), rawPtr(d.devData.az), rawPtr(d.devData.du), nidxPool, traversalPool);

    checkGpuErrors(hipGetLastError());

    float minDt;
    checkGpuErrors(hipMemcpyFromSymbol(&minDt, HIP_SYMBOL(minDt_device), sizeof(minDt)));
    d.minDtCourant = minDt;
}

template void computeMomentumEnergyStdGpu(size_t, size_t, sphexa::ParticlesData<double, unsigned, cstone::GpuTag>& d,
                                          const cstone::Box<double>&);
template void computeMomentumEnergyStdGpu(size_t, size_t, sphexa::ParticlesData<double, uint64_t, cstone::GpuTag>& d,
                                          const cstone::Box<double>&);
template void computeMomentumEnergyStdGpu(size_t, size_t, sphexa::ParticlesData<float, unsigned, cstone::GpuTag>& d,
                                          const cstone::Box<float>&);
template void computeMomentumEnergyStdGpu(size_t, size_t, sphexa::ParticlesData<float, uint64_t, cstone::GpuTag>& d,
                                          const cstone::Box<float>&);

} // namespace sph
