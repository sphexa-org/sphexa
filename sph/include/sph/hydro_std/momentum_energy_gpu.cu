#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Pressure gradient (momentum) and energy i-loop GPU driver
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "sph/sph.cuh"
#include "sph/particles_data.hpp"
#include "sph/util/cuda_utils.cuh"
#include "sph/hydro_std/momentum_energy_kern.hpp"

#include "cstone/cuda/findneighbors.cuh"

namespace sph
{
namespace cuda
{

//! @brief compute atomic min for floats using integer operations
__device__ __forceinline__ float atomicMinFloat(float* addr, float value)
{
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value)))
                       : __uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

    return old;
}

__device__ float minDt_device;

struct GradPConfig
{
    //! @brief number of threads per block for the traversal kernel
    static constexpr int numThreads = 128;
};

template<class T, class KeyType>
__global__ void cudaGradP(T sincIndex, T K, T Kcour, int ngmax, cstone::Box<T> box, int firstParticle, int lastParticle,
                          int numParticles, const KeyType* particleKeys, const T* x, const T* y, const T* z,
                          const T* vx, const T* vy, const T* vz, const T* h, const T* m, const T* rho, const T* p,
                          const T* c, const T* c11, const T* c12, const T* c13, const T* c22, const T* c23,
                          const T* c33, const T* wh, const T* whd, T* grad_P_x, T* grad_P_y, T* grad_P_z, T* du)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i   = tid + firstParticle;

    // need to hard-code ngmax stack allocation for now
    assert(ngmax <= NGMAX && "ngmax too big, please increase NGMAX to desired value");
    int neighbors[NGMAX];
    int neighborsCount;

    // starting from CUDA 11.3, dynamic stack allocation is available with the following command
    // int* neighbors = (int*)alloca(ngmax * sizeof(int));

    T dt_i = INFINITY;

    if (i < lastParticle)
    {
        cstone::findNeighbors(
            i, x, y, z, h, box, cstone::sfcKindPointer(particleKeys), neighbors, &neighborsCount, numParticles, ngmax);

        T maxvsignal;
        sph::momentumAndEnergyJLoop(i,
                                    sincIndex,
                                    K,
                                    box,
                                    neighbors,
                                    neighborsCount,
                                    x,
                                    y,
                                    z,
                                    vx,
                                    vy,
                                    vz,
                                    h,
                                    m,
                                    rho,
                                    p,
                                    c,
                                    c11,
                                    c12,
                                    c13,
                                    c22,
                                    c23,
                                    c33,
                                    wh,
                                    whd,
                                    grad_P_x,
                                    grad_P_y,
                                    grad_P_z,
                                    du,
                                    &maxvsignal);

        dt_i = sph::tsKCourant(maxvsignal, h[i], c[i], Kcour);
    }

    typedef hipcub::BlockReduce<T, GradPConfig::numThreads> BlockReduce;
    __shared__ typename BlockReduce::TempStorage         temp_storage;

    BlockReduce reduce(temp_storage);
    T           blockMin = reduce.Reduce(dt_i, hipcub::Min());
    __syncthreads();

    if (threadIdx.x == 0) { atomicMinFloat(&minDt_device, blockMin); }
}

template<class Dataset>
void computeMomentumAndEnergy(size_t startIndex, size_t endIndex, size_t ngmax, Dataset& d,
                              const cstone::Box<typename Dataset::RealType>& box)
{
    using T = typename Dataset::RealType;

    size_t sizeWithHalos = d.x.size();

    unsigned numParticlesCompute = endIndex - startIndex;

    unsigned numThreads = GradPConfig::numThreads;
    unsigned numBlocks  = (numParticlesCompute + numThreads - 1) / numThreads;

    float huge = 1e10;
    CHECK_CUDA_ERR(hipMemcpyToSymbol(HIP_SYMBOL(minDt_device), &huge, sizeof(huge)));

    cudaGradP<<<numBlocks, numThreads>>>(d.sincIndex,
                                         d.K,
                                         d.Kcour,
                                         ngmax,
                                         box,
                                         startIndex,
                                         endIndex,
                                         sizeWithHalos,
                                         rawPtr(d.devData.codes),
                                         rawPtr(d.devData.x),
                                         rawPtr(d.devData.y),
                                         rawPtr(d.devData.z),
                                         rawPtr(d.devData.vx),
                                         rawPtr(d.devData.vy),
                                         rawPtr(d.devData.vz),
                                         rawPtr(d.devData.h),
                                         rawPtr(d.devData.m),
                                         rawPtr(d.devData.rho),
                                         rawPtr(d.devData.p),
                                         rawPtr(d.devData.c),
                                         rawPtr(d.devData.c11),
                                         rawPtr(d.devData.c12),
                                         rawPtr(d.devData.c13),
                                         rawPtr(d.devData.c22),
                                         rawPtr(d.devData.c23),
                                         rawPtr(d.devData.c33),
                                         rawPtr(d.devData.wh),
                                         rawPtr(d.devData.whd),
                                         rawPtr(d.devData.ax),
                                         rawPtr(d.devData.ay),
                                         rawPtr(d.devData.az),
                                         rawPtr(d.devData.du));

    CHECK_CUDA_ERR(hipGetLastError());

    float minDt;
    CHECK_CUDA_ERR(hipMemcpyFromSymbol(&minDt, HIP_SYMBOL(minDt_device), sizeof(minDt)));
    d.minDt_loc = minDt;
}

template void computeMomentumAndEnergy(size_t, size_t, size_t,
                                       sphexa::ParticlesData<double, unsigned, cstone::GpuTag>& d,
                                       const cstone::Box<double>&);
template void computeMomentumAndEnergy(size_t, size_t, size_t,
                                       sphexa::ParticlesData<double, uint64_t, cstone::GpuTag>& d,
                                       const cstone::Box<double>&);
template void computeMomentumAndEnergy(size_t, size_t, size_t,
                                       sphexa::ParticlesData<float, unsigned, cstone::GpuTag>& d,
                                       const cstone::Box<float>&);
template void computeMomentumAndEnergy(size_t, size_t, size_t,
                                       sphexa::ParticlesData<float, uint64_t, cstone::GpuTag>& d,
                                       const cstone::Box<float>&);

} // namespace cuda
} // namespace sph
