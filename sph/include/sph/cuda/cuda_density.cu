#include "hip/hip_runtime.h"
#include "sph.cuh"

#include "sph/kernel/density_kern.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{

/*! @brief
 *
 * @tparam     T               float or double
 * @tparam     KeyType         32- or 64-bit unsigned integer
 * @param[in]  sincIndex
 * @param[in]  K
 * @param[in]  ngmax           maximum number of neighbors per particle to use
 * @param[in]  box             global coordinate bounding box
 * @param[in]  firstParticle   first particle to compute
 * @param[in]  lastParticle    last particle to compute
 * @param[in]  numParticles    number of local particles + halos
 * @param[in]  particleKeys    SFC keys of particles, sorted in ascending order
 * @param[in]  x               x coords, length @p numParticles, SFC sorted
 * @param[in]  y               y coords, length @p numParticles, SFC sorted
 * @param[in]  z               z coords, length @p numParticles, SFC sorted
 * @param[in]  h               smoothing lengths, length @p numParticles
 * @param[in]  m               masses, length @p numParticles
 * @param[in]  wh              sinc lookup table
 * @param[in]  whd             sinc derivative lookup table
 * @param[in]  rho0
 * @param[in]  wrho0
 * @param[out] rho             densities, length @p numParticles
 * @param[out] kx
 * @param[out] whomega
 *
 */
template<class T, class KeyType>
__global__ void cudaDensity(
    T sincIndex,
    T K,
    int ngmax,
    cstone::Box<T> box,
    int firstParticle,
    int lastParticle,
    int numParticles,
    const KeyType* particleKeys,
    const T* x,
    const T* y,
    const T* z,
    const T* h,
    const T* m,
    const T* wh,
    const T* whd,
    const T* rho0,
    const T* wrho0,
    T* rho,
    T* kx,
    T* whomega)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i   = tid + firstParticle;

    // need to hard-code ngmax stack allocation for now
    assert(ngmax <= NGMAX && "ngmax too big, please increase NGMAX to desired value");
    int neighbors[NGMAX];
    int neighborsCount;

    // starting from CUDA 11.3, dynamic stack allocation is available with the following command
    // int* neighbors = (int*)alloca(ngmax * sizeof(int));

    if (i < lastParticle)
    {
        cstone::findNeighbors(
            i, x, y, z, h, box, cstone::sfcKindPointer(particleKeys), neighbors, &neighborsCount, numParticles, ngmax);

        kernels::densityJLoop(i,
                              sincIndex,
                              K,
                              box,
                              neighbors,
                              neighborsCount,
                              x,
                              y,
                              z,
                              h,
                              m,
                              wh,
                              whd,
                              rho0,
                              wrho0,
                              rho,
                              kx,
                              whomega);
    }
}

template<class Dataset>
void computeDensity(size_t startIndex, size_t endIndex, size_t ngmax, Dataset& d,
                    const cstone::Box<typename Dataset::RealType>& box)
{
    using T       = typename Dataset::RealType;
    using KeyType = typename Dataset::KeyType;

    size_t sizeWithHalos     = d.x.size();
    size_t numLocalParticles = endIndex - startIndex;
    size_t size_np_T         = sizeWithHalos * sizeof(T);
    size_t size_np_CodeType  = sizeWithHalos * sizeof(KeyType);

    size_t taskSize = DeviceParticlesData<T, KeyType>::taskSize;
    size_t numTasks = iceil(numLocalParticles, taskSize);

    // number of CUDA streams to use
    constexpr int NST = DeviceParticlesData<T, Dataset>::NST;

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_codes, d.codes.data(), size_np_CodeType, hipMemcpyHostToDevice));

    for (int i = 0; i < numTasks; ++i)
    {
        int          sIdx   = i % NST;
        hipStream_t stream = d.devPtrs.d_stream[sIdx].stream;

        unsigned firstParticle       = startIndex + i * taskSize;
        unsigned lastParticle        = std::min(startIndex + (i + 1) * taskSize, endIndex);
        unsigned numParticlesCompute = lastParticle - firstParticle;

        unsigned numThreads = CudaConfig::numThreads;
        unsigned numBlocks  = (numParticlesCompute + numThreads - 1) / numThreads;

        cudaDensity<<<numBlocks, numThreads, 0, stream>>>(d.sincIndex,
                                                          d.K,
                                                          ngmax,
                                                          box,
                                                          firstParticle,
                                                          lastParticle,
                                                          sizeWithHalos,
                                                          d.devPtrs.d_codes,
                                                          d.devPtrs.d_x,
                                                          d.devPtrs.d_y,
                                                          d.devPtrs.d_z,
                                                          d.devPtrs.d_h,
                                                          d.devPtrs.d_m,
                                                          d.devPtrs.d_wh,
                                                          d.devPtrs.d_whd,
                                                          d.devPtrs.d_rho0,
                                                          d.devPtrs.d_wrho0,
                                                          d.devPtrs.d_rho,
                                                          d.devPtrs.d_kx,
                                                          d.devPtrs.d_whomega);
        CHECK_CUDA_ERR(hipGetLastError());
    }

    // Memcpy in default stream synchronizes all other streams
    CHECK_CUDA_ERR(hipMemcpy(d.rho.data(),     d.devPtrs.d_rho,     size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.kx.data(),      d.devPtrs.d_kx,      size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.whomega.data(), d.devPtrs.d_whomega, size_np_T, hipMemcpyDeviceToHost));
}

template void computeDensity(size_t, size_t, size_t, ParticlesData<double, unsigned, cstone::GpuTag>&,
                             const cstone::Box<double>&);
template void computeDensity(size_t, size_t, size_t, ParticlesData<double, uint64_t, cstone::GpuTag>&,
                             const cstone::Box<double>&);
template void computeDensity(size_t, size_t, size_t, ParticlesData<float, unsigned, cstone::GpuTag>&,
                             const cstone::Box<float>&);
template void computeDensity(size_t, size_t, size_t, ParticlesData<float, uint64_t, cstone::GpuTag>&,
                             const cstone::Box<float>&);

} // namespace cuda
} // namespace sph
} // namespace sphexa
