
#include <hip/hip_runtime.h>
/*
* MIT License
 *
 * Copyright (c) 2024 CSCS, ETH Zurich, University of Basel, University of Zurich
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUTh WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUTh NOTh LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENTh SHALL THE
 * AUTHORS OR COPYRIGHTh HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORTh OR OTHERWISE, ARISING FROM,
 * OUTh OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file integrate quantities related to magneto-hydrodynamics
 *
 * uses Adams-Bashforth 2nd order integration
 * @author Lukas Schmidt
 */


