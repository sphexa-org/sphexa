#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 17.04.2024.
//

#include "betaCooling_gpu.hpp"
#include "sph/util/device_math.cuh"
#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "cstone/sfc/box.hpp"

#include "sph/util/device_math.cuh"
#include "sph/particles_data.hpp"
#include "star_data.hpp"

#include <thrust/reduce.h>
#include <thrust/tuple.h>

#include <cmath>

namespace planet
{

template<typename Tpos, typename Tu, typename Ts, typename Tdu, typename Trho, typename Trho2>
__global__ void betaCoolingGPUKernel(size_t first, size_t last, const Tpos* x, const Tpos* y, const Tpos* z, Tdu* du,
                                     const Tu* u, Ts star_mass, Ts star_pos_x, Ts star_pos_y, Ts star_pos_z, Ts beta,
                                     Tpos g, const Trho* rho, Ts u_floor, Trho2 cooling_rho_limit)

{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= last) { return; }
    if (rho[i] >= cooling_rho_limit || u[i] <= u_floor) return;

    const double dx    = x[i] - star_pos_x;
    const double dy    = y[i] - star_pos_y;
    const double dz    = z[i] - star_pos_z;
    const double dist2 = dx * dx + dy * dy + dz * dz;
    const double dist  = sqrt(dist2);
    const double omega = sqrt(g * star_mass / (dist2 * dist));
    du[i] += -u[i] * omega / beta;
}

template<typename Dataset, typename StarData>
void betaCoolingGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    unsigned           numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    betaCoolingGPUKernel<<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.du),
        rawPtr(d.devData.u), star.m, star.position[0], star.position[1], star.position[2], star.beta, d.g,
        rawPtr(d.devData.rho), star.u_floor, star.cooling_rho_limit);

    checkGpuErrors(hipDeviceSynchronize());
}

template void betaCoolingGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);

template<typename Tu, typename Tdu>
struct AbsDivide
{
    HOST_DEVICE_FUN double operator()(const thrust::tuple<Tu, Tdu>& X)
    {
        return double{fabs(thrust::get<0>(X) / thrust::get<1>(X))};
    }
};

template<typename Dataset, typename StarData>
double duTimestepGPU(size_t first, size_t last, const Dataset& d, const StarData& star)
{
    cstone::LocalIndex numParticles = last - first;

    const auto* u  = rawPtr(d.devData.u);
    const auto* du = rawPtr(d.devData.du);

    using Tu  = std::decay_t<decltype(*u)>;
    using Tdu = std::decay_t<decltype(*du)>;

    auto begin = thrust::make_zip_iterator(u, du);
    auto end   = thrust::make_zip_iterator(u + numParticles, du + numParticles);

    double init = INFINITY;

    return star.K_u *
           thrust::transform_reduce(thrust::device, begin, end, AbsDivide<Tu, Tdu>{}, init, thrust::maximum<double>{});
}

template double duTimestepGPU(size_t, size_t, const sphexa::ParticlesData<cstone::GpuTag>&, const StarData&);

} // namespace planet
