#include "hip/hip_runtime.h"
#include <algorithm>

#include "sph.cuh"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernel/computeIAD.hpp"

#include "cstone/cuda/findneighbors.cuh"

namespace sphexa
{
namespace sph
{
namespace cuda
{

/*! @brief
 *
 * @tparam     T               float or double
 * @tparam     KeyType         32- or 64-bit unsigned integer
 * @param[in]  sincIndex
 * @param[in]  K
 * @param[in]  ngmax           maximum number of neighbors per particle to use
 * @param[in]  box             global coordinate bounding box
 * @param[in]  firstParticle   first particle to compute
 * @param[in]  lastParticle    last particle to compute
 * @param[in]  numParticles    number of local particles + halos
 * @param[in]  particleKeys    SFC keys of particles, sorted in ascending order
 * @param[in]  x               x coords, length @p numParticles, SFC sorted
 * @param[in]  y               y coords, length @p numParticles, SFC sorted
 * @param[in]  z               z coords, length @p numParticles, SFC sorted
 * @param[in]  h               smoothing lengths, length @p numParticles
 * @param[in]  m               masses, length @p numParticles
 * @param[in]  ro              densities, length @p numParticles
 * @param[in]  wh              sinc lookup table
 * @param[in]  whd             sinc derivative lookup table
 * @param[out] c11             output iad components, length @p numParticles
 * @param[out] c12
 * @param[out] c13
 * @param[out] c22
 * @param[out] c23
 * @param[out] c33
 */
template<class T, class KeyType>
__global__ void computeIAD(T sincIndex, T K, int ngmax, cstone::Box<T> box,
                           int firstParticle, int lastParticle, int numParticles, const KeyType* particleKeys,
                           const T* x, const T* y, const T* z, const T* h, const T* m, const T* ro,
                           const T* wh, const T* whd, T* c11, T* c12, T* c13, T* c22, T* c23, T* c33)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i = tid + firstParticle;

    if (i >= lastParticle) return;

    // need to hard-code ngmax stack allocation for now
    assert(ngmax <= NGMAX && "ngmax too big, please increase NGMAX to desired size");
    int neighbors[NGMAX];
    int neighborsCount;

    // starting from CUDA 11.3, dynamic stack allocation is available with the following command
    // int* neighbors = (int*)alloca(ngmax * sizeof(int));

    cstone::findNeighbors(
        i, x, y, z, h, box, cstone::sfcKindPointer(particleKeys), neighbors, &neighborsCount, numParticles, ngmax);

    sph::kernels::IADJLoop(
        i, sincIndex, K, box, neighbors, neighborsCount, x, y, z, h, m, ro, wh, whd, c11, c12, c13, c22, c23, c33);
}

template <class Dataset>
void computeIAD(const std::vector<Task>& taskList, Dataset& d, const cstone::Box<double>& box)
{
    using T = typename Dataset::RealType;

    // number of locally present particles, including halos
    size_t numParticles = d.x.size();

    size_t size_np_T = numParticles * sizeof(T);
    T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));

    unsigned firstParticle = taskList.front().firstParticle;
    unsigned lastParticle  = taskList.back().lastParticle;
    unsigned numParticlesCompute = lastParticle - firstParticle;

    unsigned numThreads = 128;
    unsigned numBlocks  = (numParticlesCompute + numThreads - 1) / numThreads;

    computeIAD<<<numBlocks, numThreads>>>(
        d.sincIndex, d.K, ngmax, box, firstParticle, lastParticle, numParticles, d.devPtrs.d_codes,
        d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, d.devPtrs.d_m, d.devPtrs.d_ro,
        d.devPtrs.d_wh, d.devPtrs.d_whd,
        d.devPtrs.d_c11, d.devPtrs.d_c12, d.devPtrs.d_c13, d.devPtrs.d_c22, d.devPtrs.d_c23, d.devPtrs.d_c33);
    CHECK_CUDA_ERR(hipGetLastError());

    CHECK_CUDA_ERR(hipMemcpy(d.c11.data(), d.devPtrs.d_c11, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c12.data(), d.devPtrs.d_c12, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c13.data(), d.devPtrs.d_c13, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c22.data(), d.devPtrs.d_c22, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c23.data(), d.devPtrs.d_c23, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c33.data(), d.devPtrs.d_c33, size_np_T, hipMemcpyDeviceToHost));
}

template void computeIAD(const std::vector<Task>& taskList, ParticlesData<double, unsigned>& d,
                         const cstone::Box<double>&);
template void computeIAD(const std::vector<Task>& taskList, ParticlesData<double, uint64_t>& d,
                         const cstone::Box<double>&);

} // namespace cuda
} // namespace sph
} // namespace sphexa

