#include "hip/hip_runtime.h"
#include <algorithm>

#include "sph.cuh"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernel/computeDensity.hpp"

#include "cstone/cuda/findneighbors.cuh"

namespace sphexa
{
namespace sph
{
namespace cuda
{

template<class T, class KeyType>
__global__ void density(T sincIndex, T K, int ngmax, cstone::Box<T> box,
                        int firstParticle, int lastParticle, int numParticles, const KeyType* particleKeys,
                        int* neighborsCount,
                        const T* x, const T* y, const T* z, const T* h, const T* m, const T* wh, const T* whd, T* ro)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned i = tid + firstParticle;
    if (i >= lastParticle) return;

    // need to hard-code ngmax stack allocation for now
    assert(ngmax <= NGMAX && "ngmax too big, please increase NGMAX to desired value");
    int neighbors[NGMAX];
    int neighborsCount_;

    // starting from CUDA 11.3, dynamic stack allocation is available with the following command
    // int* neighbors = (int*)alloca(ngmax * sizeof(int));

    cstone::findNeighbors(
        i, x, y, z, h, box, cstone::sfcKindPointer(particleKeys), neighbors, &neighborsCount_, numParticles, ngmax);

    ro[i] = sph::kernels::densityJLoop(i, sincIndex, K, box, neighbors, neighborsCount_, x, y, z, h, m, wh, whd);

    neighborsCount[tid] = neighborsCount_;
}

template<class Dataset>
void computeDensity(std::vector<Task>& taskList, Dataset& d, const cstone::Box<double>& box)
{
    using T = typename Dataset::RealType;

    size_t numParticles = d.x.size();

    size_t size_np_T = numParticles * sizeof(T);
    size_t size_np_CodeType = numParticles * sizeof(typename Dataset::KeyType);
    T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    auto largestChunkSize =
        std::max_element(taskList.cbegin(), taskList.cend(),
                         [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
            ->clist.size();

    d.devPtrs.resize_streams(largestChunkSize, ngmax);

    // number of CUDA streams to use
    constexpr int NST = DeviceParticlesData<T, Dataset>::NST;

    size_t ltsize = d.wh.size();
    size_t size_lt_T = ltsize * sizeof(T);
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_wh, d.wh.data(), size_lt_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_whd, d.whd.data(), size_lt_T, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_codes, d.codes.data(), size_np_CodeType, hipMemcpyHostToDevice));

    for (int i = 0; i < taskList.size(); ++i)
    {
        auto &t = taskList[i];

        int sIdx = i % NST;
        hipStream_t stream = d.devPtrs.d_stream[sIdx].stream;

        int* d_neighborsCount_use = d.devPtrs.d_stream[sIdx].d_neighborsCount;

        unsigned firstParticle = t.clist.front();
        unsigned lastParticle  = t.clist.back() + 1;
        unsigned numParticlesCompute = lastParticle - firstParticle;

        unsigned numThreads = 256;
        unsigned numBlocks  = (numParticlesCompute + numThreads - 1) / numThreads;

        density<<<numBlocks, numThreads, 0, stream>>>(
            d.sincIndex, d.K, t.ngmax, box,
            firstParticle, lastParticle, numParticles, d.devPtrs.d_codes, d_neighborsCount_use,
            d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, d.devPtrs.d_m, d.devPtrs.d_wh, d.devPtrs.d_whd,
            d.devPtrs.d_ro);
        CHECK_CUDA_ERR(hipGetLastError());

        CHECK_CUDA_ERR(hipMemcpyAsync(t.neighborsCount.data(), d_neighborsCount_use,
                                       numParticlesCompute * sizeof(int), hipMemcpyDeviceToHost, stream));
    }

    // Memcpy in default stream synchronizes all other streams
    CHECK_CUDA_ERR(hipMemcpy(d.ro.data(), d.devPtrs.d_ro, size_np_T, hipMemcpyDeviceToHost));

}

template void computeDensity(std::vector<Task>&, ParticlesData<double, unsigned>&, const cstone::Box<double>&);
template void computeDensity(std::vector<Task>&, ParticlesData<double, uint64_t>&, const cstone::Box<double>&);

} // namespace cuda
} // namespace sph
} // namespace sphexa

