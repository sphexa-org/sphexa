#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Find neighbors in Morton code sorted x,y,z arrays
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <iomanip>
#include <iostream>
#include <iterator>

#include <thrust/device_vector.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"

#include "cstone/traversal/find_neighbors.cuh"

#include "../coord_samples/random.hpp"
#include "timing.cuh"

using namespace cstone;

template<class T, class KeyType>
__global__ void findNeighborsKernel(const T* x,
                                    const T* y,
                                    const T* z,
                                    const T* h,
                                    LocalIndex firstId,
                                    LocalIndex lastId,
                                    const Box<T> box,
                                    const OctreeNsView<T, KeyType> treeView,
                                    unsigned ngmax,
                                    LocalIndex* neighbors,
                                    unsigned* neighborsCount)
{
    cstone::LocalIndex tid = blockDim.x * blockIdx.x + threadIdx.x;
    cstone::LocalIndex id  = firstId + tid;
    if (id >= lastId) { return; }

    findNeighbors(id, x, y, z, h, treeView, box, ngmax, neighbors + tid * ngmax, neighborsCount + id);
}

template<class T, class StrongKeyType>
void benchmarkGpu()
{
    using KeyType = typename StrongKeyType::ValueType;

    Box<T> box{0, 1, BoundaryType::periodic};
    int n = 2000000;

    RandomCoordinates<T, StrongKeyType> coords(n, box);
    std::vector<T> h(n, 0.012);

    // RandomGaussianCoordinates<T, StrongKeyType> coords(n, box);
    // adjustSmoothingLength<KeyType>(n, 100, 200, coords.x(), coords.y(), coords.z(), h, box);

    int ngmax = 200;

    std::vector<LocalIndex> neighborsCPU(ngmax * n);
    std::vector<unsigned> neighborsCountCPU(n);

    const T* x        = coords.x().data();
    const T* y        = coords.y().data();
    const T* z        = coords.z().data();
    const auto* codes = (KeyType*)(coords.particleKeys().data());

    unsigned bucketSize   = 64;
    auto [csTree, counts] = computeOctree(codes, codes + n, bucketSize);
    OctreeData<KeyType, CpuTag> octree;
    octree.resize(nNodes(csTree));
    updateInternalTree<KeyType>(csTree, octree.data());
    const TreeNodeIndex* childOffsets = octree.childOffsets.data();
    const TreeNodeIndex* toLeafOrder  = octree.internalToLeaf.data();

    std::vector<LocalIndex> layout(nNodes(csTree) + 1);
    std::exclusive_scan(counts.begin(), counts.end() + 1, layout.begin(), 0);

    std::vector<Vec3<T>> centers(octree.numNodes), sizes(octree.numNodes);
    gsl::span<const KeyType> nodeKeys(octree.prefixes.data(), octree.numNodes);
    nodeFpCenters<KeyType>(nodeKeys, centers.data(), sizes.data(), box);

    OctreeNsView<T, KeyType> nsView{octree.prefixes.data(),
                                    octree.childOffsets.data(),
                                    octree.internalToLeaf.data(),
                                    octree.levelRange.data(),
                                    layout.data(),
                                    centers.data(),
                                    sizes.data()};

    auto findNeighborsCpu = [&]()
    {
#pragma omp parallel for
        for (LocalIndex i = 0; i < n; ++i)
        {
            neighborsCountCPU[i] =
                findNeighbors(i, x, y, z, h.data(), nsView, box, ngmax, neighborsCPU.data() + i * ngmax);
        }
    };

    float cpuTime = timeCpu(findNeighborsCpu);

    std::cout << "CPU time " << cpuTime << " s" << std::endl;
    std::copy(neighborsCountCPU.data(), neighborsCountCPU.data() + 64, std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    std::vector<cstone::LocalIndex> neighborsGPU(ngmax * n);
    std::vector<unsigned> neighborsCountGPU(n);

    thrust::device_vector<T> d_x(coords.x().begin(), coords.x().end());
    thrust::device_vector<T> d_y(coords.y().begin(), coords.y().end());
    thrust::device_vector<T> d_z(coords.z().begin(), coords.z().end());
    thrust::device_vector<T> d_h = h;

    thrust::device_vector<KeyType> d_prefixes             = octree.prefixes;
    thrust::device_vector<TreeNodeIndex> d_childOffsets   = octree.childOffsets;
    thrust::device_vector<TreeNodeIndex> d_internalToLeaf = octree.internalToLeaf;
    thrust::device_vector<TreeNodeIndex> d_levelRange     = octree.levelRange;
    thrust::device_vector<LocalIndex> d_layout            = layout;
    thrust::device_vector<Vec3<T>> d_centers              = centers;
    thrust::device_vector<Vec3<T>> d_sizes                = sizes;

    OctreeNsView<T, KeyType> nsViewGpu{rawPtr(d_prefixes),   rawPtr(d_childOffsets), rawPtr(d_internalToLeaf),
                                       rawPtr(d_levelRange), rawPtr(d_layout),       rawPtr(d_centers),
                                       rawPtr(d_sizes)};

    thrust::device_vector<LocalIndex> d_neighbors(neighborsGPU.size());
    thrust::device_vector<unsigned> d_neighborsCount(neighborsCountGPU.size());

    thrust::device_vector<KeyType> d_codes(coords.particleKeys().begin(), coords.particleKeys().end());
    const auto* deviceKeys = (const KeyType*)(rawPtr(d_codes));

    auto findNeighborsLambda = [&]()
    {
        // findNeighborsKernel<<<iceil(n, 128), 128>>>(rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_h), 0, n, box,
        //                                             nsViewGpu, ngmax, rawPtr(d_neighbors), rawPtr(d_neighborsCount));

        findNeighborsBT(0, n, rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_h), nsViewGpu, box,
                        rawPtr(d_neighborsCount), rawPtr(d_neighbors), ngmax);
    };

    float gpuTime = timeGpu(findNeighborsLambda);

    thrust::copy(d_neighborsCount.begin(), d_neighborsCount.end(), neighborsCountGPU.begin());
    thrust::copy(d_neighbors.begin(), d_neighbors.end(), neighborsGPU.begin());

    std::cout << "GPU time " << gpuTime / 1000 << " s" << std::endl;
    std::copy(neighborsCountGPU.data(), neighborsCountGPU.data() + 64, std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    int numFails     = 0;
    int numFailsList = 0;
    for (int i = 0; i < n; ++i)
    {
        std::sort(neighborsCPU.data() + i * ngmax, neighborsCPU.data() + i * ngmax + neighborsCountCPU[i]);

        std::vector<cstone::LocalIndex> nilist(neighborsCountGPU[i]);
        for (unsigned j = 0; j < neighborsCountGPU[i]; ++j)
        {
            size_t warpOffset = (i / TravConfig::targetSize) * TravConfig::targetSize * ngmax;
            size_t laneOffset = i % TravConfig::targetSize;
            nilist[j]         = neighborsGPU[warpOffset + TravConfig::targetSize * j + laneOffset];
            nilist[j]         = neighborsGPU[warpOffset + TravConfig::targetSize * j + laneOffset];

            // nilist[j] = neighborsGPU[i * ngmax + j];
        }
        std::sort(nilist.begin(), nilist.end());

        if (neighborsCountGPU[i] != neighborsCountCPU[i])
        {
            std::cout << i << " " << neighborsCountGPU[i] << " " << neighborsCountCPU[i] << std::endl;
            numFails++;
        }

        if (!std::equal(begin(nilist), end(nilist), neighborsCPU.begin() + i * ngmax)) { numFailsList++; }
    }

    bool allEqual = std::equal(begin(neighborsCountGPU), end(neighborsCountGPU), begin(neighborsCountCPU));
    if (allEqual)
        std::cout << "Neighbor counts: PASS\n";
    else
        std::cout << "Neighbor counts: FAIL " << numFails << std::endl;

    std::cout << "numFailsList " << numFailsList << std::endl;
}

int main() { benchmarkGpu<double, HilbertKey<uint64_t>>(); }
